#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__constant__ int dev_N;		//Número de columnas (memoria constante)
__constant__ int dev_M;		//Número de filas (memoria constante)
__constant__ int dev_DIF;	//Dificultad (memoria constante)

int vidas = 0;				//Número de vidas
int N = 0;					//Número de columnas
int M = 0;					//Número de filas
int dif;					//Dificultad (4 -> Fácil / 6 -> Difícil)
char ejecucion;				//Tipo de ejecución (a -> Automática / m -> Manual)

//Da a cada columna una semilla para generar números aleatorios
__global__ void setup_kernel(hiprandState* state, unsigned long seed) {
	int col = (blockIdx.x * blockDim.x) + threadIdx.x;							//Columna del hilo en el tablero
	int fil = (blockIdx.y * blockDim.y) + threadIdx.y;							//Fila del hilo en el tablero
	int id = (fil * dev_N) + col;												//Posición del hilo en el tablero
	if (fil == 0)
		hiprand_init(seed, id, 0, &state[id]);									//Inicializamos la semilla (seed cambia con el tiempo)
}

//Genera fichas aleatorias en los bloques de aire que hay arriba
__global__ void generar_fichas(char* dev_tablero, hiprandState* globalState, int* dev_fichasInf) {
	int col = (blockIdx.x * blockDim.x) + threadIdx.x;							//Columna del hilo en el tablero
	int fil = (blockIdx.y * blockDim.y) + threadIdx.y;							//Fila del hilo en el tablero
	int pos = ((fil * dev_N) + col) * 2;										//Posición del hilo en el tablero

	//Si tenemos un bloque de aire en la primera fila, entonces generamos una ficha aleatoria en esa posición
	if (fil == 0 && dev_tablero[pos] == '0') {
		int idx = threadIdx.x;
		hiprandState localState = globalState[idx];								//Cogemos la semilla calculada anteriormente
		dev_tablero[pos] = (int)(hiprand_uniform(&localState) * dev_DIF) + 1;	//Obtenemos el valor aleatorio y actualizamos la semilla
		globalState[idx] = localState;											//Guardamos la semilla actualizada

		atomicSub(&dev_fichasInf[1], 1);										//Restamos 1 al número de bloques de aire
	}
}

//Baja las fichas cuando se detectan bloques de aire
__global__ void bajar_fichas(char* dev_tablero) {
	int col = (blockIdx.x * blockDim.x) + threadIdx.x;							//Columna del hilo en el tablero
	int fil = (blockIdx.y * blockDim.y) + threadIdx.y;							//Fila del hilo en el tablero
	int pos = ((fil * dev_N) + col) * 2;										//Posición del hilo en el tablero

	//Si la fila se corresponde con la última del tablero, nos recorremos la columna hacia arriba hasta encontrar bloques de aire
	if (fil == dev_M - 1) {
		for (int i = pos; i >= dev_N; i -= dev_N) {
			//Si tenemos un bloque de aire y el de arriba no lo es, tenemos que hacer que caiga la ficha
			if (dev_tablero[i * 2] == '0' && dev_tablero[(i - dev_N) * 2] != '0') {
				dev_tablero[i * 2] = dev_tablero[(i - dev_N) * 2];				//Bajamos la ficha
				dev_tablero[i * 2 + 1] = dev_tablero[(i - dev_N) * 2 + 1];
				dev_tablero[(i - dev_N) * 2] = '0';								//Ponemos el bloque de aire en la posición de encima
				dev_tablero[(i - dev_N) * 2 + 1] = '0';
			}
		}
	}

}

//Elimina las fichas (la que se toca el usuario y las equivalentes que están juntas)
__global__ void eliminar_fichas(char* dev_tablero, int* dev_coordenadas, int* dev_fichaInf, hiprandState* globalState) {
	int col = blockIdx.x * blockDim.x + threadIdx.x;							//Columna del hilo en el tablero
	int fil = blockIdx.y * blockDim.y + threadIdx.y;							//Fila del hilo en el tablero
	int idx = (fil * blockDim.x + col) * 2;										//Posición del hilo en el tablero
	int touch = (dev_coordenadas[1] * blockDim.x + dev_coordenadas[0]) * 2;		//Posición del elemento que ha tocado el usuario
	char elem = dev_tablero[touch];												//Elemento que ha tocado el usuario

	if (idx == touch) {
		dev_tablero[idx] = '0';													//Si el hilo actual es el que ha tocado el usuario, ponemos un bloque de aire
	}
	else if (dev_tablero[idx] == elem) {
		atomicAdd(&dev_fichaInf[0], 1);											//Si el elemento actual coincide con el que ha tocado el usuario, sumamos 1 al contador
	}

	__syncthreads();															//Sincronizamos los hilos
	int total = dev_fichaInf[0];												//Número de fichas que coinciden con el elemento que ha tocado el usuario
	dev_fichaInf[1] = dev_fichaInf[0];											//Igualamos los contadores que controlan el bucle (si son iguales es que no se ha eliminado ninguna ficha)
	bool encontrado = false;													//Variable de control para llevar a cabo el borrado
	do {
		dev_fichaInf[0] = dev_fichaInf[1];										//Igualamos los contadores en cada iteración
		__syncthreads();														//Sincronizamos los hilos
		if (dev_tablero[idx] == elem && dev_tablero[idx] != '0') {				//Si el elemento del hilo coincide con el elemento que ha tocado el usuario y no ha sido borrado, comprobamos si se puede borrar
			if (col + 1 < blockDim.x && dev_tablero[idx + 2] == '0') {
				encontrado = true;												//Si a la derecha tiene un bloque de aire, ha encontrado una ruta al bloque que ha tocado el usuario y debe ser borrado
			}
			else if (col - 1 >= 0 && dev_tablero[idx - 2] == '0') {
				encontrado = true;												//Si a la izquierda tiene un bloque de aire, ha encontrado una ruta al bloque que ha tocado el usuario y debe ser borrado
			}
			else if (fil + 1 < blockDim.y && dev_tablero[idx + (blockDim.x * 2)] == '0') {
				encontrado = true;												//Si abajo tiene un bloque de aire, ha encontrado una ruta al bloque que ha tocado el usuario y debe ser borrado
			}
			else if (fil - 1 >= 0 && dev_tablero[idx - (blockDim.x * 2)] == '0') {
				encontrado = true;												//Si arriba tiene un bloque de aire, ha encontrado una ruta al bloque que ha tocado el usuario y debe ser borrado
			}
			if (encontrado) {													//Si se ha encontrado una ruta hasta el bloque que ha tocado el usuario, borramos el bloque
				atomicSub(&dev_fichaInf[1], 1);									//Restamos 1 al contador
				dev_tablero[idx] = '0';											//Colocamos un bloque de aire
			}
		}
		__syncthreads();														//Sincronizamos los hilos
	} while (dev_fichaInf[0] != dev_fichaInf[1]);								//El bucle continua mientras los contadores no sean iguales (si se ha borrado algún elemento)

	if (idx == touch) {															//Si el bloque coincide con el que ha tocado el usuario, comprobamos el número de fichas que se eliminan para colocar los bloques especiales
		dev_fichaInf[1] = total - dev_fichaInf[1] + 1;							//Número de fichas eliminadas
		if (dev_fichaInf[1] == 5) {												//Si se han eliminado 5, colocamos una bomba (B)
			dev_tablero[idx] = 'B';
			dev_fichaInf[1] -= 1;												//Restamos 1 a los bloques que se han borrado (creado aire)
		}
		else if (dev_fichaInf[1] == 6) {										//Si se han eliminado 6, colocamos una TNT (T)
			dev_tablero[idx] = 'T';
			dev_fichaInf[1] -= 1;												//Restamos 1 a los bloques que se han borrado (creado aire)
		}
		else if (dev_fichaInf[1] >= 7) {										//Si se han eliminado 7 o más, colocamos un rompecabezas (R)
			dev_tablero[idx] = 'R';
			int id = threadIdx.x;
			hiprandState localState = globalState[id];							//Seleccionamos aleatoriamente el tipo de rompecabezas
			dev_tablero[idx + 1] = (int) (hiprand_uniform(&localState) * dev_DIF) + 1;
			globalState[id] = localState;
			dev_fichaInf[1] -= 1;												//Restamos 1 a los bloques que se han borrado (creado aire)
		}
	}
}

//Elimina la bomba (borra la fila o columna aleatoriamente)
__global__ void eliminar_bomba(char* dev_tablero, int* dev_coordenadas, int* dev_fichaInf, int aleatorio) {
	int pos = (threadIdx.y * blockDim.x + threadIdx.x) * 2;						//Posición del hilo en el tablero

	if (aleatorio == 0) {														//Si aleatorio es 0, comprueba que coincide la columna con la del bloque que ha tocado el usuario
		if (threadIdx.y == dev_coordenadas[1]) {
			dev_tablero[pos] = '0';												//Se borran todos los elementos de la columna
			atomicAdd(&dev_fichaInf[1], 1);
		}
	}
	else {																		//Si aleatorio es 1, comprueba que coincide la fila con la del bloque que ha tocado el usuario
		if (threadIdx.x == dev_coordenadas[0]) {
			dev_tablero[pos] = '0';												//Se borran todos los elementos de la fila
			atomicAdd(&dev_fichaInf[1], 1);										//Suma 1 a los bloques que se han convertido en aire
		}
	}
}

//Elimina la TNT (borra los bloques adyacentes en un radio de 4)
__global__ void eliminar_tnt(char* dev_tablero, int* dev_coordenadas, int* dev_fichaInf) {
	int pos = (threadIdx.y * blockDim.x + threadIdx.x) * 2;						//Posición del hilo en el tablero
	int distancia = sqrt((float)pow((double)threadIdx.x - dev_coordenadas[0], 2.0) + pow((double)threadIdx.y - dev_coordenadas[1], 2.0));		//Calculamos la distancia entre dos puntos

	if (distancia <= 4) {														//Si el bloque está en un radio de 4, lo borra
		dev_tablero[pos] = '0';
		atomicAdd(&dev_fichaInf[1], 1);											//Suma 1 a los bloques que se han convertido en aire
	}
}

//Elimina el rompecabezas (borra los bloques cuyo elemento coincida con el del rompecabezas)
__global__ void eliminar_rompecabezas(char* dev_tablero, int* dev_coordenadas, int* dev_fichaInf) {
	int pos = (threadIdx.y * blockDim.x + threadIdx.x) * 2;						//Posición del hilo en el tablero
	int touch = (dev_coordenadas[1] * blockDim.x + dev_coordenadas[0]) * 2;		//Posición del bloque que ha tocado el usuario
	char elem = dev_tablero[touch + 1];											//Obtenemos el elemento que hay en el bloque que ha tocado el usuario

	__syncthreads();															//Sincronizamos los hilos

	if (dev_tablero[pos] == elem || pos == touch) {								//Si el elemento coincide con el que ha tocado el usuario o si es el que ha tocado el usuario, lo borra
		dev_tablero[pos] = '0';													//Borra el elemento
		atomicAdd(&dev_fichaInf[1], 1);											//Suma 1 a los bloques que se han convertido en aire
	}
}

void update_tablero() {

}

void run_game() {

}

//Rellena con bloques de aire todo el tablero
void vaciar_tablero(char* tablero) {
	for (int i = 0; i < N * M * 2; i++) {
		tablero[i] = '0';
	}
}

//Imprime el tablero por pantalla
void mostrar_tablero(char* tablero) {
	for (int i = 0; i < M; i++) {												//Recorremos las filas
		printf("\n\n| ");
		for (int j = 0; j < N; j++) {											//Recorremos las columnas
			if ((int)tablero[((i * N) + j) * 2] > 0 && (int)tablero[((i * N) + j) * 2] <= 6)
				printf("%d | ", tablero[((i * N) + j) * 2]);					//Si es una ficha

			else if (tablero[((i * N) + j) * 2] == 'R')							//Si es un rompecabezas
				printf("%c%d | ", tablero[((i * N) + j) * 2], tablero[((i * N) + j) * 2 + 1]);

			else
				printf("%c | ", tablero[((i * N) + j) * 2]);					//Si es otro bloque especial
		}
		printf("\n");
		for (int j = 0; j < N; j++) {
			printf(" ---");
		}
	}
	printf("\n-----------------------------------------------------------\n");
}

//Main
int main(int argc, const char* argv[]) {
	hipFree(0);

	//Datos usuario
	vidas = 100;
	N = 11;				//columnas
	M = 6;				//filas
	dif = 6;
	ejecucion = 'm';

	//Declaración de variables
	int SIZE = N * M * 2 * sizeof(char);
	int size_coord = 2 * sizeof(int);
	int* h_coordenadas = (int*)malloc(size_coord);								//Coordenadas del bloque que toca el usuario
	int* h_fichaInf = (int*)malloc(size_coord);									//Información de las fichas (número de bloques eliminados / número de fichas del mismo tipo que ha pulsado el usuario)
	char* h_tablero = (char*)malloc(SIZE);										//Tablero

	//Variables GPU
	hiprandState* dev_states;													//Guarda las semillas para los números aleatorios
	char* dev_tablero;															//Tablero
	int* dev_coordenadas;														//Coordenadas del bloque que toca el usuario
	int* dev_fichaInf;															//Información de las fichas (número de fichas del mismo tipo que ha pulsado el usuario / número de bloques eliminados)

	hipMemcpyToSymbol(HIP_SYMBOL(dev_DIF), &dif, sizeof(int));								//Pasa dif a dev_DIF (memoria constante)
	hipMemcpyToSymbol(HIP_SYMBOL(dev_N), &N, sizeof(int));									//Pasa N a dev_N (memoria constante)
	hipMemcpyToSymbol(HIP_SYMBOL(dev_M), &M, sizeof(int));									//Pasa M a dev_M (memoria constante)
	hipMalloc(&dev_states, N * sizeof(hiprandState));							//Reserva memoria global para dev_states
	hipMalloc((void**)&dev_coordenadas, size_coord);							//Reserva memoria global para dev_coordenadas
	hipMalloc((void**)&dev_fichaInf, size_coord);								//Reserva memoria global para dev_fichaInf
	hipMalloc((void**)&dev_tablero, SIZE);										//Reserva memoria global para dev_tablero

	//Inicializar tablero
	//---------------------------------------------------------------------------------------------
	h_fichaInf[0] = 0;															//El usuario no ha tocado ningún bloque
	h_fichaInf[1] = N * M;														//Todas las casillas son bloques de aire

	vaciar_tablero(h_tablero);													//Vaciamos el tablero
	mostrar_tablero(h_tablero);													//Mostramos el tablero

	hipMemcpy(dev_tablero, h_tablero, SIZE, hipMemcpyHostToDevice);			//Transfiere el contenido de h_tablero a dev_tablero
	hipMemcpy(dev_fichaInf, h_fichaInf, size_coord, hipMemcpyHostToDevice);	//Transfiere el contenido de h_fichaInf a dev_fichaInf
	dim3 blocksInGrid(1);														//1 bloque
	dim3 threadsInBlock(N, M);													//N * M hilos
	setup_kernel << <blocksInGrid, threadsInBlock >> > (dev_states, time(0));	//Generamos las semillas

	while (h_fichaInf[1] != 0) {												//Se llama iterativamente a bajar_fichas y generar_fichas hasta que no queden bloques de aire
		bajar_fichas << <blocksInGrid, threadsInBlock >> > (dev_tablero);
		generar_fichas << <blocksInGrid, threadsInBlock >> > (dev_tablero, dev_states, dev_fichaInf);

		hipMemcpy(h_tablero, dev_tablero, SIZE, hipMemcpyDeviceToHost);
		mostrar_tablero(h_tablero);												//Mostramos el tablero
		hipMemcpy(h_fichaInf, dev_fichaInf, size_coord, hipMemcpyDeviceToHost);
	}

	//Bucle principal
	//---------------------------------------------------------------------------------------------
	int ficha;
	srand(time(NULL));

	printf("\nComienza el juego :)\n");

	//Mientras tengamos vidas, el juego continúa
	while (vidas > 0) {
		printf("\nVidas: %d\n", vidas);
		if (ejecucion == 'm') {													//Ejecución manual
			printf("Ejecucion manual <m>\n");
			do {																//Solicitamos las coordenadas al usuario
				printf("Introduce el numero de columna (%d - %d): ", 0, N - 1);
				scanf("%d", &h_coordenadas[0]);
			} while (h_coordenadas[0] < 0 || h_coordenadas[0] >= N);

			do {
				printf("Introduce el numero de fila (%d - %d): ", 0, M - 1);
				scanf("%d", &h_coordenadas[1]);
			} while (h_coordenadas[1] < 0 || h_coordenadas[1] >= M);
		}
		else {																	//Ejecución automática
			printf("Ejecucion automatica <a>\n");
			h_coordenadas[0] = rand() % N;										//Seleccionamos las coordenadas aleatoriamente
			h_coordenadas[1] = rand() % M;
		}
		printf("Coordenadas: (%d, %d)\n", h_coordenadas[0], h_coordenadas[1]);

		ficha = (h_coordenadas[1] * N + h_coordenadas[0]) * 2;					//Posición del bloque con la coordenadas

		hipMemcpy(dev_coordenadas, h_coordenadas, size_coord, hipMemcpyHostToDevice);

		if ((int)h_tablero[ficha] <= 6) {										//Eliminamos fichas
			eliminar_fichas << <blocksInGrid, threadsInBlock >> > (dev_tablero, dev_coordenadas, dev_fichaInf, dev_states);
		}
		else if (h_tablero[ficha] == 'B') {										//Eliminamos bomba
			int aleatorio = rand() % 2;
			eliminar_bomba << <blocksInGrid, threadsInBlock >> > (dev_tablero, dev_coordenadas, dev_fichaInf, aleatorio);
		}
		else if (h_tablero[ficha] == 'T') {										//Eliminamos TNT
			eliminar_tnt << <blocksInGrid, threadsInBlock >> > (dev_tablero, dev_coordenadas, dev_fichaInf);
		}
		else {																	//Eliminamos rompecabezas
			eliminar_rompecabezas << <blocksInGrid, threadsInBlock >> > (dev_tablero, dev_coordenadas, dev_fichaInf);
		}
		hipMemcpy(h_tablero, dev_tablero, SIZE, hipMemcpyDeviceToHost);
		hipMemcpy(h_fichaInf, dev_fichaInf, size_coord, hipMemcpyDeviceToHost);
		mostrar_tablero(h_tablero);												//Mostramos el tablero

		if (h_fichaInf[1] == 1) vidas--;										//Restamos una vida si se ha eliminado sólamente un bloque

		while (h_fichaInf[1] != 0) {											//Se llama iterativamente a bajar_fichas y generar_fichas hasta que no queden bloques de aire
			bajar_fichas << <blocksInGrid, threadsInBlock >> > (dev_tablero);
			generar_fichas << <blocksInGrid, threadsInBlock >> > (dev_tablero, dev_states, dev_fichaInf);

			hipMemcpy(h_tablero, dev_tablero, SIZE, hipMemcpyDeviceToHost);
			mostrar_tablero(h_tablero);											//Mostramos el tablero
			hipMemcpy(h_fichaInf, dev_fichaInf, size_coord, hipMemcpyDeviceToHost);
		}

	}
	printf("\nVidas: %d\n", vidas);
	printf("\nGAME OVER :(\n");

	//Liberar memoria
	//---------------------------------------------------------------------------------------------
	hipFree(dev_tablero);
	hipFree(dev_coordenadas);
	hipFree(dev_fichaInf);
	hipFree(dev_states);
	free(h_tablero);
	free(h_coordenadas);
	free(h_fichaInf);

}
