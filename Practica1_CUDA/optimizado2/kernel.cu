#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__constant__ int dev_N;		//N�mero de columnas (memoria constante)
__constant__ int dev_M;		//N�mero de filas (memoria constante)
__constant__ int dev_DIF;	//Dificultad (memoria constante)

int vidas = 0;				//N�mero de vidas
int N = 0;					//N�mero de columnas
int M = 0;					//N�mero de filas
int dif;					//Dificultad (4 -> F�cil / 6 -> Dif�cil)
char ejecucion;				//Tipo de ejecuci�n (a -> Autom�tica / m -> Manual)

int BLQ_X;
int BLQ_Y;
const int TESELA_X = 6;
const int TESELA_Y = 6;

//Da a cada columna una semilla para generar n�meros aleatorios
__global__ void setup_kernel(hiprandState* state, unsigned long seed) {
	int col = (blockIdx.x * blockDim.x) + threadIdx.x;									//Columna del hilo en el tablero
	int fil = (blockIdx.y * blockDim.y) + threadIdx.y;									//Fila del hilo en el tablero
	int id = (fil * dev_N) + col;														//Posici�n del hilo en el tablero
	if (dev_N > col && dev_M > fil) {													//Comprobamos si estamos dentro del bloque
		if (fil == 0)
			hiprand_init(seed, id, 0, &state[id]);										//Inicializamos la semilla (seed cambia con el tiempo)
	}
}

//Genera fichas aleatorias en los bloques de aire que hay arriba
__global__ void generar_fichas(char* dev_tablero, hiprandState* globalState, int* dev_fichasInf) {
	__shared__ char tab_shared[TESELA_X * TESELA_Y * 2];								//reservamos memoria compartida
	int pos_shared = (threadIdx.y * blockDim.x + threadIdx.x) * 2;						//obtenemos la pocision de memoria compartida para el hilo

	int col = (blockIdx.x * blockDim.x) + threadIdx.x;									//Columna del hilo en el tablero
	int fil = (blockIdx.y * blockDim.y) + threadIdx.y;									//Fila del hilo en el tablero
	int pos = ((fil * dev_N) + col) * 2;												//Posici�n del hilo en el tablero

	if (dev_N > col && dev_M > fil) {													//Comprobamos si estamos dentro del bloque
		tab_shared[pos_shared] = dev_tablero[pos];										//Cargamos el tablero en memoria compartida
		tab_shared[pos_shared + 1] = dev_tablero[pos + 1];	
	}
	else {
		tab_shared[pos_shared] = '0';													//Lo que sobra se rellena con 0's
		tab_shared[pos_shared + 1] = '0';
	}
	__syncthreads();

	//Si tenemos un bloque de aire en la primera fila, entonces generamos una ficha aleatoria en esa posici�n
	if (dev_N > col && dev_M > fil) {													//Comprobamos si estamos dentro del bloque
		if (fil == 0 && tab_shared[pos_shared] == '0') {

			int idx = threadIdx.x;
			hiprandState localState = globalState[idx];									//Cogemos la semilla calculada anteriormente
			tab_shared[pos_shared] = (int)(hiprand_uniform(&localState) * dev_DIF) + 1;	//Obtenemos el valor aleatorio y actualizamos la semilla
			globalState[idx] = localState;												//Guardamos la semilla actualizada
			atomicSub(&dev_fichasInf[1], 1);											//Restamos 1 al n�mero de bloques de aire
		}
	}

	__syncthreads();

	if (dev_N > col && dev_M > fil)														
		dev_tablero[pos] = tab_shared[pos_shared];										//Se actualiza el tablero con los cambios realizados en memoria compartida
}

//Baja las fichas cuando se detectan bloques de aire
__global__ void bajar_fichas(char* dev_tablero) {
	int col = (blockIdx.x * blockDim.x) + threadIdx.x;									//Columna del hilo en el tablero
	int fil = (blockIdx.y * blockDim.y) + threadIdx.y;									//Fila del hilo en el tablero
	int pos = ((fil * dev_N) + col) * 2;												//Posici�n del hilo en el tablero

	//Si la fila se corresponde con la �ltima del tablero, nos recorremos la columna hacia arriba hasta encontrar bloques de aire
	if (dev_N > col && dev_M > fil) {
		if (fil == dev_M - 1) {
			for (int i = pos; i >= dev_N * 2; i -= dev_N * 2) {
				//Si tenemos un bloque de aire y el de arriba no lo es, tenemos que hacer que caiga la ficha
				if (dev_tablero[i] == '0' && dev_tablero[(i - dev_N * 2)] != '0') {
					dev_tablero[i] = dev_tablero[(i - dev_N * 2)];						//Bajamos la ficha
					dev_tablero[i + 1] = dev_tablero[(i - dev_N * 2) + 1];
					dev_tablero[(i - dev_N * 2)] = '0';									//Ponemos el bloque de aire en la posici�n de encima
					dev_tablero[(i - dev_N * 2) + 1] = '0';
				}
			}
		}
	}
}

//Elimina las fichas (la que se toca el usuario y las equivalentes que est�n juntas)
__global__ void eliminar_fichas(char* dev_tablero, int* dev_coordenadas, int* dev_fichaInf) {
	__shared__ char tab_shared[TESELA_X * TESELA_Y * 2];								//reservamos memoria compartida
	int pos_shared = (threadIdx.y * blockDim.x + threadIdx.x) * 2;						//obtenemos la pocision de memoria compartida para el hilo

	int col = blockIdx.x * blockDim.x + threadIdx.x;									//Columna del hilo en el tablero
	int fil = blockIdx.y * blockDim.y + threadIdx.y;									//Fila del hilo en el tablero
	int idx = (fil * dev_N + col) * 2;													//Posici�n del hilo en el tablero
	char elem = dev_fichaInf[2];
	bool encontrado = false;

	if (dev_N > col && dev_M > fil) {													//Comprobamos si estamos dentro del bloque
		tab_shared[pos_shared] = dev_tablero[idx];										//Cargamos el tablero en memoria compartida
		tab_shared[pos_shared + 1] = dev_tablero[idx + 1];
	}
	else {
		tab_shared[pos_shared] = 'F';													//Lo que sobra se rellena con 'F's
		tab_shared[pos_shared + 1] = 'F';
	}
	__syncthreads();

	if (dev_N > col && dev_M > fil && tab_shared[pos_shared] == elem && tab_shared[pos_shared] != '0') {	//Comprobamos si estamos dentro del bloque y si somos candidatos a eliminar
		//mira derecha
		if (!encontrado && col + 1 < dev_N) {											//compobamos que no nos salimos al ir a la derecha del tablero
			if (threadIdx.x + 1 < TESELA_X) {											//comprobamos si estamos en los limites de la tesela
				if (tab_shared[pos_shared + 2] == '0')									//si encontramos el elemento 0
					encontrado = true;													//encontramos camino
			}
			else {																		//si nos salimos de la tesela
				if (dev_tablero[idx + 2] == '0')										//buscamos camino en memoria global
					encontrado = true;													//encontramos camino
			}
		}
		//mira izquierda
		if (!encontrado && col - 1 >= 0) {												//compobamos que no nos salimos al ir a la izquierda del tablero
			if ((int)threadIdx.x - 1 >= 0) {											//comprobamos si estamos en los limites de la tesela
				if (tab_shared[pos_shared - 2] == '0')									//si encontramos el elemento 0
					encontrado = true;													//encontramos camino
			}
			else {																		//si nos salimos de la tesela
				if (dev_tablero[idx - 2] == '0')										//buscamos camino en memoria global
					encontrado = true;													//encontramos camino
			}	
		}
		//mira abajo
		if (!encontrado && fil + 1 < dev_M) {											//compobamos que no nos salimos al ir hacia abajo en el tablero
			if (threadIdx.y + 1 < TESELA_Y) {											//comprobamos si estamos en los limites de la tesela
				if (tab_shared[pos_shared + (TESELA_X * 2)] == '0')						//si encontramos el elemento 0
					encontrado = true;													//encontramos camino
			}
			else {																		//si nos salimos de la tesela
				if (dev_tablero[idx + (dev_N * 2)] == '0')								//buscamos camino en memoria global
					encontrado = true;													//encontramos camino
			}
		}
		//mira arriba
		if (!encontrado && fil - 1 >= 0) {												//compobamos que no nos salimos al ir hacia arriba en el tablero
			if ((int)threadIdx.y - 1 >= 0) {											//comprobamos si estamos en los limites de la tesela
				if (tab_shared[pos_shared - (TESELA_X * 2)] == '0')						//si encontramos el elemento 0
					encontrado = true;													//encontramos camino
			}
			else {																		//si nos salimos de la tesela
				if (dev_tablero[idx - (dev_N * 2)] == '0')								//buscamos camino en memoria global
					encontrado = true;													//encontramos camino
			}
		}

		if (encontrado) {																//Si se ha encontrado una ruta hasta el bloque que ha tocado el usuario, borramos el bloque
			tab_shared[pos_shared] = '0';												//Colocamos un bloque de aire
			atomicAdd(&dev_fichaInf[1], 1);												//Aumenta el contador de fichas eliminadas
			atomicSub(&dev_fichaInf[0], 1);												//Restamos 1 al contador de fichas del tipo que se quiere eliminar
		}
	}
	__syncthreads();

	if (dev_N > col && dev_M > fil) {
		dev_tablero[idx] = tab_shared[pos_shared];										//Se actualiza el tablero con los cambios realizados en memoria compartida
		dev_tablero[idx + 1] = tab_shared[pos_shared + 1];
	}

}

//Coloca el punto de busqueda de eliminar fichas y fichas especiales
__global__ void colocar_fichaEX(char* dev_tablero, int* dev_coordenadas, int* dev_fichaInf, hiprandState* globalState) {
	__shared__ char tab_shared[TESELA_X * TESELA_Y * 2];								//reservamos memoria compartida
	int pos_shared = (threadIdx.y * blockDim.x + threadIdx.x) * 2;						//obtenemos la pocision de memoria compartida para el hilo
		
	int col = blockIdx.x * blockDim.x + threadIdx.x;									//Columna del hilo en el tablero
	int fil = blockIdx.y * blockDim.y + threadIdx.y;									//Fila del hilo en el tablero
	int idx = (fil * dev_N + col) * 2;													//Posici�n del hilo en el tablero
	int touch = (dev_coordenadas[1] * dev_N + dev_coordenadas[0]) * 2;

	if (dev_N > col && dev_M > fil) {													//Comprobamos si estamos dentro del bloque
		tab_shared[pos_shared] = dev_tablero[idx];										//Cargamos el tablero en memoria compartida
		tab_shared[pos_shared + 1] = dev_tablero[idx + 1];
	}
	else {
		tab_shared[pos_shared] = '0';													//Lo que sobra se rellena con '0's
		tab_shared[pos_shared + 1] = '0';
	}
	__syncthreads();
		
	if (dev_N > col && dev_M > fil) {													//Comprobamos si estamos dentro del bloque
		if (idx == touch) {																//Comprobamos que somos el hilo de las coordenadas del usuario
			if (dev_fichaInf[1] == 1) {													//Si solo se ha eliminado 1
				tab_shared[pos_shared] = dev_fichaInf[2];								//Volvemos a la ficha anterior
				dev_fichaInf[1] -= 1;
			}
			else if (dev_fichaInf[1] == 0) {											//Si aun no se ha eliminado ninguna
				dev_fichaInf[2] = tab_shared[pos_shared];								//Colocamos el punto de busqueda para eliminar fichas
				tab_shared[pos_shared] = '0';
				dev_fichaInf[1] += 1;
			}
			else if (dev_fichaInf[1] == 5) {											//Si se han eliminado 5 fichas
				tab_shared[pos_shared] = 'B';											//Se coloca una bomba en el tablero
				dev_fichaInf[1] -= 1;
			}
			else if (dev_fichaInf[1] == 6) {											//Si se han eliminado 6 fichas
				tab_shared[pos_shared] = 'T';											//Se coloca una TNT en el talbero
				dev_fichaInf[1] -= 1;
			}
			else if (dev_fichaInf[1] >= 7) {											//Si se han eliminado 7 fichas
				tab_shared[pos_shared] = 'R';											//Se coloca un tompecabezas en el tablero
				int id = col;
				hiprandState localState = globalState[id];
				tab_shared[pos_shared + 1] = (int)(hiprand_uniform(&localState) * dev_DIF) + 1;	//Seleccionamos el complemento del rompecabezas de forma aleatoria
				globalState[id] = localState;
				dev_fichaInf[1] -= 1;
			}
		}
		else {
			atomicAdd(&dev_fichaInf[0], 1);
		}
	}
	__syncthreads();

	if (dev_N > col && dev_M > fil) {
		dev_tablero[idx] = tab_shared[pos_shared];										//Se actualiza el tablero con los cambios realizados en memoria compartida
		dev_tablero[idx + 1] = tab_shared[pos_shared + 1];
	}
}

//Elimina la bomba (borra la fila o columna aleatoriamente)
__global__ void eliminar_bomba(char* dev_tablero, int* dev_coordenadas, int* dev_fichaInf, int aleatorio) {
	__shared__ char tab_shared[TESELA_X * TESELA_Y * 2];								//reservamos memoria compartida
	int pos_shared = (threadIdx.y * blockDim.x + threadIdx.x) * 2;						//obtenemos la pocision de memoria compartida para el hilo

	int col = blockIdx.x * blockDim.x + threadIdx.x;									//Columna del hilo en el tablero
	int fil = blockIdx.y * blockDim.y + threadIdx.y;									//Fila del hilo en el tablero
	int pos = (fil * dev_N + col) * 2;													//Posici�n del hilo en el tablero

	if (dev_N > col && dev_M > fil) {													//Comprobamos si estamos dentro del bloque
		tab_shared[pos_shared] = dev_tablero[pos];										//Cargamos el tablero en memoria compartida
		tab_shared[pos_shared + 1] = dev_tablero[pos + 1];
	}
	else {
		tab_shared[pos_shared] = '0';													//Lo que sobra se rellena con '0's
		tab_shared[pos_shared + 1] = '0';
	}
	__syncthreads();

	if (dev_N > col && dev_M > fil) {													//Comprobamos si estamos dentro del bloque
		if (aleatorio == 0) {															//Si aleatorio es 0, comprueba que coincide la columna con la del bloque que ha tocado el usuario
			if (fil == dev_coordenadas[1]) {
				tab_shared[pos_shared] = '0';											//Se borran todos los elementos de la columna
				atomicAdd(&dev_fichaInf[1], 1);											
			}
		}
		else {																			//Si aleatorio es 1, comprueba que coincide la fila con la del bloque que ha tocado el usuario
			if (col == dev_coordenadas[0]) {
				tab_shared[pos_shared] = '0';											//Se borran todos los elementos de la fila
				atomicAdd(&dev_fichaInf[1], 1);											//Suma 1 a los bloques que se han convertido en aire
			}
		}
	}
	__syncthreads();

	if (dev_N > col && dev_M > fil) {
		dev_tablero[pos] = tab_shared[pos_shared];										//Se actualiza el tablero con los cambios realizados en memoria compartida
		dev_tablero[pos + 1] = tab_shared[pos_shared + 1];
	}
}

//Elimina la TNT (borra los bloques adyacentes en un radio de 4)
__global__ void eliminar_tnt(char* dev_tablero, int* dev_coordenadas, int* dev_fichaInf) {
	__shared__ char tab_shared[TESELA_X * TESELA_Y * 2];								//reservamos memoria compartida
	int pos_shared = (threadIdx.y * blockDim.x + threadIdx.x) * 2;						//obtenemos la pocision de memoria compartida para el hilo

	int col = blockIdx.x * blockDim.x + threadIdx.x;									//Columna del hilo en el tablero
	int fil = blockIdx.y * blockDim.y + threadIdx.y;									//Fila del hilo en el tablero
	int pos = (fil * dev_N + col) * 2;													//Posici�n del hilo en el tablero
	int distancia = sqrt((float)pow((double)col - dev_coordenadas[0], 2.0) + pow((double)fil - dev_coordenadas[1], 2.0));	//Calculamos la distancia entre dos puntos

	if (dev_N > col && dev_M > fil) {													//Comprobamos si estamos dentro del bloque
		tab_shared[pos_shared] = dev_tablero[pos];										//Cargamos el tablero en memoria compartida
		tab_shared[pos_shared + 1] = dev_tablero[pos + 1];
	}
	else {
		tab_shared[pos_shared] = '0';													//Lo que sobra se rellena con '0's
		tab_shared[pos_shared + 1] = '0';
	}
	__syncthreads();

	if (dev_N > col && dev_M > fil) {
		if (distancia <= 4) {															//Si el bloque est� en un radio de 4, lo borra
			tab_shared[pos_shared] = '0';
			atomicAdd(&dev_fichaInf[1], 1);												//Suma 1 a los bloques que se han convertido en aire
		}	
	}
	__syncthreads();

	if (dev_N > col && dev_M > fil) {
		dev_tablero[pos] = tab_shared[pos_shared];										//Se actualiza el tablero con los cambios realizados en memoria compartida
		dev_tablero[pos + 1] = tab_shared[pos_shared + 1];
	}
}

//Elimina el rompecabezas (borra los bloques cuyo elemento coincida con el del rompecabezas)
__global__ void eliminar_rompecabezas(char* dev_tablero, int* dev_coordenadas, int* dev_fichaInf) {
	__shared__ char tab_shared[TESELA_X * TESELA_Y * 2];								//reservamos memoria compartida
	int pos_shared = (threadIdx.y * blockDim.x + threadIdx.x) * 2;						//obtenemos la pocision de memoria compartida para el hilo
		
	int col = blockIdx.x * blockDim.x + threadIdx.x;									//Columna del hilo en el tablero
	int fil = blockIdx.y * blockDim.y + threadIdx.y;									//Fila del hilo en el tablero
	int pos = (fil * dev_N + col) * 2;													//Posici�n del hilo en el tablero
	int touch = (dev_coordenadas[1] * dev_N + dev_coordenadas[0]) * 2;
	char elem = dev_tablero[touch + 1];													//Obtenemos el elemento del rompecabezas a eliminar

	if (dev_N > col && dev_M > fil) {													//Comprobamos si estamos dentro del bloque
		tab_shared[pos_shared] = dev_tablero[pos];										//Cargamos el tablero en memoria compartida
		tab_shared[pos_shared + 1] = dev_tablero[pos + 1];
	}
	else {
		tab_shared[pos_shared] = '0';													//Lo que sobra se rellena con '0's
		tab_shared[pos_shared + 1] = '0';
	}
	__syncthreads();

	__syncthreads();
	if (dev_N > col && dev_M > fil) {													//Comprobamos si estamos dentro del bloque
		if (tab_shared[pos_shared] == elem || pos == touch) {							//Si el elemento coincide con el que ha tocado el usuario o si es el que ha tocado el usuario,
			tab_shared[pos_shared] = '0';												//Borra el elemento
			atomicAdd(&dev_fichaInf[1], 1);												//Suma 1 a los bloques que se han convertido en aire
		}
	}
	__syncthreads();

	if (dev_N > col && dev_M > fil) {
		dev_tablero[pos] = tab_shared[pos_shared];										//Se actualiza el tablero con los cambios realizados en memoria compartida
		dev_tablero[pos + 1] = tab_shared[pos_shared + 1];
	}
}

void update_tablero() {

}

void run_game() {

}

//Rellena con bloques de aire todo el tablero
void vaciar_tablero(char* tablero) {
	for (int i = 0; i < N * M * 2; i++) {
		tablero[i] = '0';
	}
}

//Imprime el tablero por pantalla
void mostrar_tablero(char* tablero) {
	for (int i = 0; i < M; i++) {														//Recorremos las filas	
		printf("\n\n| ");
		for (int j = 0; j < N; j++) {													//Recorremos las columnas
			if ((int)tablero[((i * N) + j) * 2] > 0 && (int)tablero[((i * N) + j) * 2] <= 6)
				printf("%d | ", tablero[((i * N) + j) * 2]);							//Si es una ficha

			else if (tablero[((i * N) + j) * 2] == 'R')									//Si es un rompecabezas
				printf("%c%d | ", tablero[((i * N) + j) * 2], tablero[((i * N) + j) * 2 + 1]);

			else
				printf("%c | ", tablero[((i * N) + j) * 2]);							//Si es otro bloque especial
		}
		printf("\n");
		for (int j = 0; j < N; j++) {
			printf(" ---");
		}
	}
	printf("\n-----------------------------------------------------------\n");
}

int main(int argc, const char* argv[]) {
	hipFree(0);

	//Datos usuario
	vidas = 5;

	//Pedir datos al usuario por comando
	if (argc > 1) {
		if (argc != 5) {
			return -1;
		}
		printf("Hasta aqui si\n");
		N = atoi(argv[3]);
		M = atoi(argv[4]);

		printf("sigue aqui si %d -- %d\n", N, M);
		if (argv[2][0] == '1') {
			dif = 4;
		}
		else {
			dif = 6;
		}

		if (argv[1][1] == 'a') {
			ejecucion = 'a';
		}
		else {
			ejecucion = 'm';
		}
	}
	else {
		//Pedir datos al usuario por consola
		do {
			printf("Introduce el numero de filas del tablero: ");
			scanf("%d", &M);
		} while ((int)M < 1);

		do {
			printf("Introduce el numero de columnas del tablero: ");
			scanf("%d", &N);
		} while ((int)N < 1);

		do {
			printf("Introduce el tipo de ejecucion (m --> Manual / a --> Automatica): ");
			scanf("%c", &ejecucion);
		} while (ejecucion != 'm' && ejecucion != 'a');

		do {
			printf("Introduce la dificultad del juego (1 --> Facil / 2 --> Dificil): ");
			scanf("%d", &dif);
		} while (dif != 1 && dif != 2);

		if (dif == 1) dif = 4;
		else dif = 6;
	}

	//Optimizar dimensiones
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	printf("\nNombre GPU: %s\n", deviceProp.name);
	int BLOCK_SIZE = sqrt(deviceProp.maxThreadsPerBlock);
	printf("\nBlock size: %d\n", BLOCK_SIZE);

	BLQ_X = ceil((float)N / TESELA_X);
	BLQ_Y = ceil((float)M / TESELA_Y);

	dim3 blocksInGrid(BLQ_X, BLQ_Y);
	dim3 threadsInBlock(TESELA_X, TESELA_Y);
	printf("TESELA: %d, %d -- DIM:%d, %d --> tam %d, %d", TESELA_X, TESELA_Y , N, M, BLQ_X, BLQ_Y);

	//Declaraci�n de variables
	int SIZE = N * M * 2 * sizeof(char);
	int size_coord = 2 * sizeof(int);
	int size_ficha = 3 * sizeof(int);
	int* h_coordenadas = (int*)malloc(size_coord);									//Coordenadas del bloque que toca el usuario
	int* h_fichaInf = (int*)malloc(size_ficha);										//Informaci�n de las fichas (n�mero de bloques eliminados / n�mero de fichas del mismo tipo que ha pulsado el usuario)
	char* h_tablero = (char*)malloc(SIZE);											//Tablero

	//Variables GPU
	hiprandState* dev_states;														//Guarda las semillas para los n�meros aleatorios
	char* dev_tablero;																//Tablero
	int* dev_coordenadas;															//Coordenadas del bloque que toca el usuario
	int* dev_fichaInf;																//Informaci�n de las fichas (n�mero de fichas del mismo tipo que ha pulsado el usuario / n�mero de bloques eliminados)

	hipMemcpyToSymbol(HIP_SYMBOL(dev_DIF), &dif, sizeof(int));									//Pasa dif a dev_DIF (memoria constante)
	hipMemcpyToSymbol(HIP_SYMBOL(dev_N), &N, sizeof(int));										//Pasa N a dev_N (memoria constante)
	hipMemcpyToSymbol(HIP_SYMBOL(dev_M), &M, sizeof(int));										//Pasa M a dev_M (memoria constante)
	hipMalloc(&dev_states, N * sizeof(hiprandState));								//Reserva memoria global para dev_states
	hipMalloc((void**)&dev_coordenadas, size_coord);								//Reserva memoria global para dev_coordenadas
	hipMalloc((void**)&dev_fichaInf, size_ficha);									//Reserva memoria global para dev_fichaInf
	hipMalloc((void**)&dev_tablero, SIZE);											//Reserva memoria global para 

	//Inicializar tablero
	//---------------------------------------------------------------------------------------------
	h_fichaInf[0] = 0;																//El usuario no ha tocado ning�n bloque
	h_fichaInf[1] = N * M;															//Todas las casillas son bloques de aire

	vaciar_tablero(h_tablero);														//Vaciamos el tablero
	mostrar_tablero(h_tablero);														//Mostramos el tablero

	hipMemcpy(dev_tablero, h_tablero, SIZE, hipMemcpyHostToDevice);				//Transfiere el contenido de h_tablero a dev_tablero
	hipMemcpy(dev_fichaInf, h_fichaInf, size_ficha, hipMemcpyHostToDevice);		//Transfiere el contenido de h_fichaInf a dev_fichaInf

	setup_kernel << <blocksInGrid, threadsInBlock >> > (dev_states, time(0));		//Generamos las semillas

	while (h_fichaInf[1] != 0) {													//Se llama iterativamente a bajar_fichas y generar_fichas hasta que no queden bloques de aire
		bajar_fichas << <blocksInGrid, threadsInBlock >> > (dev_tablero);
		generar_fichas << <blocksInGrid, threadsInBlock >> > (dev_tablero, dev_states, dev_fichaInf);
		hipMemcpy(h_tablero, dev_tablero, SIZE, hipMemcpyDeviceToHost);
		mostrar_tablero(h_tablero);													//Mostramos el tablero
		hipMemcpy(h_fichaInf, dev_fichaInf, size_ficha, hipMemcpyDeviceToHost);
	}

	//Bucle principal
	//---------------------------------------------------------------------------------------------
	int ficha;
	srand(time(NULL));

	printf("\nComienza el juego :)\n");
	while (vidas > 0) {
		printf("\nVidas: %d\n", vidas);
		if (ejecucion == 'm') {
			printf("Ejecucion manual <m>\n");										//Ejecuci�n manual
			do {																	//Solicitamos las coordenadas al usuario
				printf("Introduce el numero de columna (%d - %d): ", 0, N - 1);
				scanf("%d", &h_coordenadas[0]);
			} while (h_coordenadas[0] < 0 || h_coordenadas[0] >= N);

			do {
				printf("Introduce el numero de fila (%d - %d): ", 0, M - 1);
				scanf("%d", &h_coordenadas[1]);
			} while (h_coordenadas[1] < 0 || h_coordenadas[1] >= M);
		}
		else {																		//Ejecuci�n autom�tica
			printf("Ejecucion automatica <a>\n");									
			h_coordenadas[0] = rand() % N;											//Seleccionamos las coordenadas aleatoriamente
			h_coordenadas[1] = rand() % M;
		}
		printf("Coordenadas: (%d, %d)\n", h_coordenadas[0], h_coordenadas[1]);

		ficha = (h_coordenadas[1] * N + h_coordenadas[0]) * 2;						//Posici�n del bloque con la coordenadas

		hipMemcpy(dev_coordenadas, h_coordenadas, size_coord, hipMemcpyHostToDevice);

		if ((int)h_tablero[ficha] <= 6) {											//Eliminamos fichas
			colocar_fichaEX << <blocksInGrid, threadsInBlock >> > (dev_tablero, dev_coordenadas, dev_fichaInf, dev_states);		//Colocamos la ficha a eliminar
			hipMemcpy(h_fichaInf, dev_fichaInf, size_ficha, hipMemcpyDeviceToHost);
			int salir;
			do {																	//Eliminamos fichas del tablero 
				salir = h_fichaInf[0];
				eliminar_fichas << <blocksInGrid, threadsInBlock >> > (dev_tablero, dev_coordenadas, dev_fichaInf);
				hipMemcpy(h_fichaInf, dev_fichaInf, size_ficha, hipMemcpyDeviceToHost);
			} while (salir != h_fichaInf[0]);

			colocar_fichaEX << <blocksInGrid, threadsInBlock >> > (dev_tablero, dev_coordenadas, dev_fichaInf, dev_states);		//Sustituimos la ficha tocada por su equivalente de fichas eliminadas
		}
		else if (h_tablero[ficha] == 'B') {											//Eliminamos bomba
			int aleatorio = rand() % 2;
			eliminar_bomba << <blocksInGrid, threadsInBlock >> > (dev_tablero, dev_coordenadas, dev_fichaInf, aleatorio);
		}
		else if (h_tablero[ficha] == 'T') {											//Eliminamos TNT
			eliminar_tnt << <blocksInGrid, threadsInBlock >> > (dev_tablero, dev_coordenadas, dev_fichaInf);
		}
		else {																		//Eliminamos rompecabezas
			eliminar_rompecabezas << <blocksInGrid, threadsInBlock >> > (dev_tablero, dev_coordenadas, dev_fichaInf);
		}
		hipMemcpy(h_tablero, dev_tablero, SIZE, hipMemcpyDeviceToHost);
		hipMemcpy(h_fichaInf, dev_fichaInf, size_ficha, hipMemcpyDeviceToHost);
		mostrar_tablero(h_tablero);													//Mostramos el tablero

		printf("\nEliminadas: %d\n", h_fichaInf[1]);
		if (h_fichaInf[1] == 0) vidas--;											//Restamos una vida si se ha eliminado s�lamente un bloque	
		
		while (h_fichaInf[1] > 0) {													//Se llama iterativamente a bajar_fichas y generar_fichas hasta que no queden bloques de aire
			bajar_fichas << <blocksInGrid, threadsInBlock >> > (dev_tablero);
			generar_fichas << <blocksInGrid, threadsInBlock >> > (dev_tablero, dev_states, dev_fichaInf);
			hipMemcpy(h_tablero, dev_tablero, SIZE, hipMemcpyDeviceToHost);
			mostrar_tablero(h_tablero);												//Mostramos el tablero
			hipMemcpy(h_fichaInf, dev_fichaInf, size_ficha, hipMemcpyDeviceToHost);
		}

		h_fichaInf[0] = 0;															//Reiniciamos las fichas que podrian borrarse
		hipMemcpy(dev_fichaInf, h_fichaInf, size_ficha, hipMemcpyHostToDevice);
	}
	printf("\nVidas: %d\n", vidas);
	printf("\nGAME OVER :(\n");

	//Liberar memoria
	//---------------------------------------------------------------------------------------------
	hipFree(dev_tablero);
	hipFree(dev_coordenadas);
	hipFree(dev_fichaInf);
	hipFree(dev_states);
	free(h_tablero);
	free(h_coordenadas);
	free(h_fichaInf);

}
